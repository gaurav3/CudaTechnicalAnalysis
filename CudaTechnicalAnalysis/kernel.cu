
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t MovAvgWithCuda(float *result, const float *input, size_t size);

__global__ void MovAvgKernel(float *result, const float *input)
{
    int i = threadIdx.x;
    result[i] = input[i];
}

int main()
{
    const int arraySize = 10000;

	const int avgWindowSize = 15;

	float a[arraySize] = {0};
	for(int i=0; i<arraySize; i++)
	{
		a[i] = i;
	}

    float result[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = MovAvgWithCuda(result, a, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("AllGood");
	getchar();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t MovAvgWithCuda(float *result, const float *input, size_t size)
{
	const int BLOCKS = 1;
	const int THREADS = 256;

    float *dev_input = 0;
	int dev_avgWindowSize = 0;
    float *dev_result = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_result, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_input, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    MovAvgKernel<<<1, 256>>>(dev_result, dev_input);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(result, dev_result, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_result);
    hipFree(dev_input);
    
    return cudaStatus;
}
